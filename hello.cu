
#include <hip/hip_runtime.h>
#include <stdio.h>

// Kernel function to add the elements of two vectors
__global__ void vectorAdd(const float* A, const float* B, float* C, int n)
{
    // Get the thread index
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    // Make sure we don't go out of bounds
    if (idx < n)
    {
        C[idx] = A[idx] + B[idx];
    }
}

int main()
{
    const int n = 100000; // Size of the vectors
    const int size = n * sizeof(float);

    // Allocate memory on the host
    float* h_A = (float*)malloc(size);
    float* h_B = (float*)malloc(size);
    float* h_C = (float*)malloc(size);

    // Initialize the vectors on the host
    for (int i = 0; i < n; ++i)
    {
        h_A[i] = 1.0f;
        h_B[i] = 2.0f;
    }

    // Allocate memory on the device
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Copy data from host to device
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Set the number of threads and blocks
    const int blockSize = 256;
    const int numBlocks = (n + blockSize - 1) / blockSize;

    // Launch the kernel
    vectorAdd<<<numBlocks, blockSize>>>(d_A, d_B, d_C, n);

    // Copy the result from device to host
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Print the first 10 elements of the result
    printf("Result:\n");
    for (int i = 0; i < 10; ++i)
    {
        printf("%f ", h_C[i]);
    }
    printf("\n");

    // Free memory on the host and device
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
